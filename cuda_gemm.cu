#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cassert>
#include "mkl.h"
#include "utils.h"

#define SINGLE_PRECISION //Comment out to use double precision arithmetic
#define DOUBLE_PRECISION

#ifdef SINGLE_PRECISION
	#define elem_t float
	#define blasGemm cblas_sgemm 
	#define cublasGemm hipblasSgemm
#elif defined(DOUBLE_PRECISION)
	#define elem_t double
	#define blasGemm cblas_dgemm 
	#define cublasGemm hipblasDgemm
#endif

#ifndef GEMM_M
#define GEMM_M 256
#endif
#ifndef GEMM_N
#define GEMM_N 256
#endif
#ifndef GEMM_K
#define GEMM_K 256
#endif

#ifndef WARMUPS
#define WARMUPS 3
#endif
#ifndef ITERS
#define ITERS 10
#endif

__global__ void gemmV1(int M, int N, int K, elem_t alpha, elem_t *A, int ldA, elem_t *B, int ldB, elem_t beta, elem_t *C, int ldC)
{
	//int row_id = threadIdx.y + blockIdx.y * blockDim.y;
        //int col_id = threadIdx.x + blockIdx.x * blockDim.x;
	//x is innermost i.e. two threads with same y are contiguous => contiguous memory access
	int row_id = threadIdx.x + blockIdx.x * blockDim.x;
        int col_id = threadIdx.y + blockIdx.y * blockDim.y;

	if (row_id >= M || col_id >= N)
		return;

	if (alpha == 0.0)
	{
		C[row_id + col_id*ldC] *= beta;
		return;
	}

	elem_t result = 0.0;
	for (int k=0; k<K; k++)
		result += A[row_id + k*ldA]*B[k + col_id*ldB];
	C[row_id + col_id*ldC] = alpha * result + beta * C[row_id + col_id*ldC];
}

//We change and play with this
#define V2_TILE_M 32
#define V2_TILE_N 32
#define V2_TILE_K 16


__global__ void gemmV2(int M, int N, int K, elem_t alpha, elem_t *A, int ldA, elem_t *B, int ldB, elem_t beta, elem_t *C, int ldC)
{
        int row_id = threadIdx.x + blockIdx.x * blockDim.x;
	int col_id = threadIdx.y + blockIdx.y * blockDim.y;

	if (row_id >= M || col_id >= N)
		return;

	if (alpha == 0.0)
	{
		C[row_id + col_id*ldC] *= beta;
		return;
	}

	assert(blockDim.x == V2_TILE_M);
	assert(blockDim.y == V2_TILE_N);
	__shared__ elem_t tile_A[V2_TILE_M*V2_TILE_K];
	__shared__ elem_t tile_B[V2_TILE_K*V2_TILE_N];

	//starting positions
	A += blockDim.x * blockIdx.x;
	B += blockDim.y * blockIdx.y * ldB;

	elem_t result = 0.0; //still one thread = one result
	for (int k=0; k<K; k+=V2_TILE_K)
	{
		//First load tile of A and B in shared memory
		//we have V2_TILE_M x V2_TILE_N threads and we need to load V2_TILE_M x V2_TILE_K elements of A
		for (int col = threadIdx.y; col < V2_TILE_K; col += blockDim.y)
			tile_A[threadIdx.x + col*V2_TILE_M] = A[threadIdx.x + col*ldA];
		//we have V2_TILE_M x V2_TILE_N threads and we need to load V2_TILE_K x V2_TILE_N elements of B
		for (int row = threadIdx.x; row < V2_TILE_K; row += blockDim.x)
			tile_B[row + threadIdx.y*V2_TILE_K] = B[row + threadIdx.y*ldB];

		A += V2_TILE_K * ldA;
		B += V2_TILE_K;

		__syncthreads(); //synchro before reading the tiles

		//Compute product of two tiles and accumulate
		for (int ik=0; ik<V2_TILE_K; ik++)
			result += tile_A[threadIdx.x + ik*V2_TILE_M]*tile_B[ik + threadIdx.y * V2_TILE_K];

		__syncthreads(); //more synchro before writing the tiles 
	}
	C[row_id + col_id*ldC] = alpha * result + beta * C[row_id + col_id*ldC];
}

#define V3_TILE_M 64
#define V3_TILE_N 64
#define V3_TILE_K 8
#define V3_THREAD_M 2
#define V3_THREAD_N 2
 void gemmV3(int M, int N, int K, elem_t alpha, elem_t *A, int ldA, elem_t *B, int ldB, elem_t beta, elem_t *C, int ldC)
{
	//write kernel with shared memory and higher arithmetic intensity
}

void runGemmV1(int M, int N, int K, elem_t alpha, elem_t *A, int ldA, elem_t *B, int ldB, elem_t beta, elem_t *C, int ldC)
{
	int threadsM = 16;
	int threadsN = 16;
	dim3 blockSize(threadsM, threadsN); //threadblock of threadsM*threadsN threads
	//We need at least M*N threads to fully compute C
	int blocksM = (M+threadsM-1) / threadsM;
	int blocksN = (N+threadsN-1) / threadsN;
	dim3 gridSize(blocksM, blocksN);
	gemmV1<<<gridSize, blockSize>>>(M,N,K,alpha,A,ldA,B,ldB,beta,C,ldC);
}
void runGemmV2(int M, int N, int K, elem_t alpha, elem_t *A, int ldA, elem_t *B, int ldB, elem_t beta, elem_t *C, int ldC)
{
	int threadsM = V2_TILE_M;
	int threadsN = V2_TILE_N;
	dim3 blockSize(threadsM, threadsN);
	int blocksM = (M+threadsM-1) / threadsM;
	int blocksN = (N+threadsN-1) / threadsN;
	dim3 gridSize(blocksM, blocksN);
	gemmV2<<<gridSize, blockSize>>>(M,N,K,alpha,A,ldA,B,ldB,beta,C,ldC);
}
void runGemmV3(int M, int N, int K, elem_t alpha, elem_t *A, int ldA, elem_t *B, int ldB, elem_t beta, elem_t *C, int ldC)
{
	//call the gpu kernel
}

int main(int argc, char **argv)
{
	float *times = new float[2*ITERS];
	float *timesCPU = times;
	float *timesGPU = times + ITERS;

	elem_t *A, *B, *C, *Cgpu;
	int M = GEMM_M;
	int N = GEMM_N;
	int K = GEMM_K;
	allocateMatrixCPU(M,K,&A);
	allocateMatrixCPU(K,N,&B);
	allocateMatrixCPU(M,N,&C);

	initMatrixRandomCPU<elem_t>(M,K,A);
	initMatrixRandomCPU<elem_t>(K,N,B);
	initMatrixCPU<elem_t>(M,N,C,0.0);

	elem_t *d_A, *d_B, *d_C;
	allocateMatrixGPU(M,K,&d_A);
	allocateMatrixGPU(K,N,&d_B);
	allocateMatrixGPU(M,N,&d_C);

	hipMemcpy(d_A, A, sizeof(elem_t)*M*K, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(elem_t)*N*K, hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, sizeof(elem_t)*M*N, hipMemcpyHostToDevice);

	elem_t alpha = 1.0;
	elem_t beta = 0.0;

	//CPU
	struct timespec cpu_start, cpu_end;
	for (int i=0; i<ITERS; i++)
	{
		clock_gettime(CLOCK_MONOTONIC, &cpu_start);
		blasGemm(CblasColMajor, CblasNoTrans, CblasNoTrans, M, N, K, alpha, A, M, B, K, beta, C, M);
		clock_gettime(CLOCK_MONOTONIC, &cpu_end);
		timesCPU[i] = computeCPUTime(&cpu_start, &cpu_end);
	}

	//GPU
	for (int i=0; i<WARMUPS; i++)
	{
		//runGemmV1(M, N, K, alpha, d_A, M, d_B, K, beta, d_C, M);
		runGemmV2(M, N, K, alpha, d_A, M, d_B, K, beta, d_C, M);
		//runGemmV3(M, N, K, alpha, d_A, M, d_B, K, beta, d_C, M);
		hipDeviceSynchronize();
	}
	hipEvent_t gpu_start, gpu_end;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_end);
	for (int i=0; i<ITERS; i++)
	{
		hipEventRecord(gpu_start);
		//runGemmV1(M, N, K, alpha, d_A, M, d_B, K, beta, d_C, M);
		runGemmV2(M, N, K, alpha, d_A, M, d_B, K, beta, d_C, M);
		//runGemmV3(M, N, K, alpha, d_A, M, d_B, K, beta, d_C, M);
		hipEventRecord(gpu_end);
		hipDeviceSynchronize();
		hipEventElapsedTime(&(timesGPU[i]), gpu_start, gpu_end);
	}
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_end);

	float flops = 2*(float)M*(float)N*(float)K;

	float avg_cpu=0.0;
	for (int i=0; i<ITERS; i++)
		avg_cpu += timesCPU[i];
	avg_cpu = avg_cpu / (float)ITERS;
	std::cout << "==== CPU ====\n";
	std::cout << "Execution time: " << avg_cpu << " ms.\n";
	std::cout << "Performance: " << (flops/1.0e9)/(avg_cpu/1.0e3) << " GFLOP/s.\n";

	float avg_gpu=0.0;
	for (int i=0; i<ITERS; i++)
		avg_gpu += timesGPU[i];
	avg_gpu = avg_gpu / (float)ITERS;
	std::cout << "==== GPU ====\n";
	std::cout << "Execution time: " << avg_gpu << " ms.\n";
	std::cout << "Performance: " << (flops/1.0e9)/(avg_gpu/1.0e3) << " GFLOP/s.\n";

	allocateMatrixCPU(M,N,&Cgpu);
	hipMemcpy(Cgpu, d_C, sizeof(elem_t)*M*N, hipMemcpyDeviceToHost);
	std::cout << std::setprecision(10);
	compareMatrices(M,N,C,Cgpu);
	freeMatrixCPU(M,N,Cgpu);

	freeMatrixGPU(M,K,d_A);
	freeMatrixGPU(K,N,d_B);
	freeMatrixGPU(M,N,d_C);

	freeMatrixCPU(M,K,A);
	freeMatrixCPU(K,N,B);
	freeMatrixCPU(M,N,C);

	delete[] times;

}
